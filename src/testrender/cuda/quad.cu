#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/AcademySoftwareFoundation/OpenShadingLanguage


#include <optix.h>

#include "rend_lib.h"
#include "render_params.h"
#include "wrapper.h"


extern "C" __device__ void
__direct_callable__quad_shaderglobals(const unsigned int idx, const float t_hit,
                                      const float3 ray_origin,
                                      const float3 ray_direction,
                                      ShaderGlobals* sg)
{
    const GenericData* g_data = reinterpret_cast<const GenericData*>(
        optixGetSbtDataPointer());
    const QuadParams* g_quads = reinterpret_cast<const QuadParams*>(
        g_data->data);
    const QuadParams& quad = g_quads[idx];
    const float3 P         = ray_origin + t_hit * ray_direction;

    float3 h = P - quad.p;

    sg->N = sg->Ng  = quad.n;
    sg->u           = dot(h, quad.ex) * quad.eu;
    sg->v           = dot(h, quad.ey) * quad.ev;
    sg->dPdu        = quad.ey;
    sg->dPdv        = quad.ex;
    sg->surfacearea = quad.a;
    sg->shaderID    = quad.shaderID;
}


extern "C" __global__ void
__intersection__quad()
{
    const GenericData* g_data = reinterpret_cast<const GenericData*>(
        optixGetSbtDataPointer());
    const QuadParams* g_quads = reinterpret_cast<const QuadParams*>(
        g_data->data);
    const unsigned int idx     = optixGetPrimitiveIndex();
    const QuadParams& quad     = g_quads[idx];
    const float3 ray_origin    = optixGetObjectRayOrigin();
    const float3 ray_direction = optixGetObjectRayDirection();

    float dn = dot(ray_direction, quad.n);
    float en = dot(quad.p - ray_origin, quad.n);
    if (dn * en > 0) {
        float t  = en / dn;
        float3 h = (ray_origin + ray_direction * t) - quad.p;
        float dx = dot(h, quad.ex) * quad.eu;
        float dy = dot(h, quad.ey) * quad.ev;

        if (dx >= 0 && dx < 1.0f && dy >= 0 && dy < 1.0f
            && t < optixGetRayTmax())
            optixReportIntersection(t, RAYTRACER_HIT_QUAD);
    }
}

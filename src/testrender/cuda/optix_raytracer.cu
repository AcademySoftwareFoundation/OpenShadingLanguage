#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/AcademySoftwareFoundation/OpenShadingLanguage


#include <optix.h>

#include "util.h"

#include <optix_device.h>

#include <OSL/device_string.h>

#include "rend_lib.h"
#include "render_params.h"


extern "C" {
__device__ __constant__ RenderParams render_params;
}


extern "C" __global__ void
__miss__()
{
    uint3 launch_dims  = optixGetLaunchDimensions();
    uint3 launch_index = optixGetLaunchIndex();

    float3* output_buffer = reinterpret_cast<float3*>(
        render_params.output_buffer);

    int pixel            = launch_index.y * launch_dims.x + launch_index.x;
    output_buffer[pixel] = make_float3(0, 0, 1);
}


extern __device__ char* test_str_1;
extern __device__ char* test_str_2;


extern "C" __global__ void
__raygen__setglobals()
{
    // Set global variables
    OSL::pvt::osl_printf_buffer_start = render_params.osl_printf_buffer_start;
    OSL::pvt::osl_printf_buffer_end   = render_params.osl_printf_buffer_end;
    OSL::pvt::s_color_system          = render_params.color_system;
    OSL::pvt::test_str_1              = render_params.test_str_1;
    OSL::pvt::test_str_2              = render_params.test_str_2;
}



extern "C" __global__ void
__miss__setglobals()
{
}



extern "C" __global__ void
__raygen__()
{
    uint3 launch_dims  = optixGetLaunchDimensions();
    uint3 launch_index = optixGetLaunchIndex();
    const float3 eye   = render_params.eye;
    const float3 dir   = render_params.dir;
    const float3 cx    = render_params.cx;
    const float3 cy    = render_params.cy;
    const float invw   = render_params.invw;
    const float invh   = render_params.invh;

    // Compute the pixel coordinates
    const float2 d = make_float2(static_cast<float>(launch_index.x) + 0.5f,
                                 static_cast<float>(launch_index.y) + 0.5f);

    // Make the ray for the current pixel
    RayGeometry r;
    r.origin    = eye;
    r.direction = normalize(cx * (d.x * invw - 0.5f) + cy * (0.5f - d.y * invh)
                            + dir);
    optixTrace(render_params.traversal_handle, r.origin, r.direction, 1e-3f,
               1e13f, 0, OptixVisibilityMask(1), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
               0, 1, 0);
}


// Because clang++ 9.0 seems to have trouble with some of the texturing "intrinsics"
// let's do the texture look-ups in this file.
extern "C" __device__ float4
osl_tex2DLookup(void* handle, float s, float t)
{
    hipTextureObject_t texID = hipTextureObject_t(handle);
    return tex2D<float4>(texID, s, t);
}

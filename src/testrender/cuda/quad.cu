#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/AcademySoftwareFoundation/OpenShadingLanguage


#include <optix.h>

#include "optix_raytracer.h"
#include "rend_lib.h"
#include "vec_math.h"


extern "C" __device__ void
__direct_callable__quad_shaderglobals(const unsigned int idx, const float t_hit,
                                      const float3 ray_origin,
                                      const float3 ray_direction,
                                      OSL_CUDA::ShaderGlobals* sg)
{
    const GenericData* g_data = reinterpret_cast<const GenericData*>(
        optixGetSbtDataPointer());
    const QuadParams* g_quads = reinterpret_cast<const QuadParams*>(
        g_data->data);
    const QuadParams& quad = g_quads[idx];
    const float3 P         = ray_origin + t_hit * ray_direction;

    float3 h = P - quad.p;

    sg->I = ray_direction;
    sg->N = sg->Ng  = quad.n;
    sg->u           = dot(h, quad.ex) * quad.eu;
    sg->v           = dot(h, quad.ey) * quad.ev;
    sg->dPdu        = quad.ey;
    sg->dPdv        = quad.ex;
    sg->surfacearea = quad.a;
    sg->shaderID    = quad.shaderID;
    sg->backfacing  = dot(V3_TO_F3(sg->N), V3_TO_F3(sg->I)) > 0.0f;

    if (sg->backfacing) {
        sg->N  = -sg->N;
        sg->Ng = -sg->Ng;
    }
}


extern "C" __global__ void
__intersection__quad()
{
    const GenericData* g_data = reinterpret_cast<const GenericData*>(
        optixGetSbtDataPointer());
    const QuadParams* g_quads = reinterpret_cast<const QuadParams*>(
        g_data->data);

    Payload payload;
    payload.get();
    OSL_CUDA::ShaderGlobals* sg_ptr = (OSL_CUDA::ShaderGlobals*)payload.sg_ptr;
    TraceData* tracedata   = reinterpret_cast<TraceData*>(sg_ptr->tracedata);
    const int obj_id       = tracedata->obj_id;
    const unsigned int idx = optixGetPrimitiveIndex();
    const QuadParams& quad = g_quads[idx];

    // Check for self-intersection
    const bool self = obj_id == quad.objID;
    if (self) {
        return;
    }

    const float3 ray_origin    = optixGetObjectRayOrigin();
    const float3 ray_direction = optixGetObjectRayDirection();
    float dn                   = dot(ray_direction, quad.n);
    float en                   = dot(quad.p - ray_origin, quad.n);
    if (dn * en > 0) {
        float t  = en / dn;
        float3 h = (ray_origin + ray_direction * t) - quad.p;
        float dx = dot(h, quad.ex) * quad.eu;
        float dy = dot(h, quad.ey) * quad.ev;

        if (dx >= 0 && dx < 1.0f && dy >= 0 && dy < 1.0f
            && t < optixGetRayTmax())
            optixReportIntersection(t, RAYTRACER_HIT_QUAD);
    }
}

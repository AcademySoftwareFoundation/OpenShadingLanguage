#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/imageworks/OpenShadingLanguage


#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_vector_types.h>


using namespace optix;

rtDeclareVariable (float3, p,  , );
rtDeclareVariable (float3, ex, , );
rtDeclareVariable (float3, ey, , );
rtDeclareVariable (float3, n,  , );
rtDeclareVariable (float,  eu, , );
rtDeclareVariable (float,  ev, , );
rtDeclareVariable (float,  a, ,  );

rtDeclareVariable (float3, texcoord,         attribute texcoord, );
rtDeclareVariable (float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable (float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable (float,  surface_area,     attribute surface_area, );

rtDeclareVariable (float3, dPdu, attribute dPdu, );
rtDeclareVariable (float3, dPdv, attribute dPdv, );

rtDeclareVariable (optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect (void)
{
    float dn = dot(ray.direction, n);
    float en = dot(p - ray.origin, n);
    if (dn * en > 0) {
        float  t  = en / dn;
        float3 h  = (ray.origin + ray.direction * t) - p;
        float  dx = dot(h, ex) * eu;
        float  dy = dot(h, ey) * ev;

        if (dx >= 0 && dx < 1.0f && dy >= 0 && dy < 1.0f && rtPotentialIntersection(t)) {
            shading_normal = geometric_normal = n;
            texcoord = make_float3(dot (h, ex) * eu, dot (h, ey) * ev, 0.0f);
            dPdu = ey;
            dPdv = ex;
            surface_area = a;
            rtReportIntersection(0);
        }
    }
}


RT_PROGRAM void bounds (int, float result[6])
{
    const float3 p00  = p;
    const float3 p01  = p + ex;
    const float3 p10  = p + ey;
    const float3 p11  = p + ex + ey;
    const float  area = length(cross(ex, ey));

    optix::Aabb* aabb = reinterpret_cast<optix::Aabb*>(result);

    if (area > 0.0f && !isinf(area)) {
        aabb->m_min = fminf (fminf (p00, p01), fminf (p10, p11));
        aabb->m_max = fmaxf (fmaxf (p00, p01), fmaxf (p10, p11));
    } else {
        aabb->invalidate();
    }
}

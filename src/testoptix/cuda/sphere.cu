#include "hip/hip_runtime.h"
/*
Copyright (c) 2009-2018 Sony Pictures Imageworks Inc., et al.
All Rights Reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:
* Redistributions of source code must retain the above copyright
  notice, this list of conditions and the following disclaimer.
* Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions and the following disclaimer in the
  documentation and/or other materials provided with the distribution.
* Neither the name of Sony Pictures Imageworks nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_vector_types.h>

#include <OSL/oslconfig.h>
#include <OSL/shaderglobals.h>
#include <OSL/dual.h>
#include <OSL/dual_vec.h>
#include <OpenImageIO/fmath.h>


using namespace optix;
using OSL::Dual2;
using OSL::Vec3;


rtDeclareVariable (float4, sphere, , );
rtDeclareVariable (float,  r2, , );
rtDeclareVariable (float,  a, , );

rtDeclareVariable (float3, texcoord,         attribute texcoord, );
rtDeclareVariable (float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable (float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable (float,  surface_area,     attribute surface_area, );

rtDeclareVariable (float3, dPdu, attribute dPdu, );
rtDeclareVariable (float3, dPdv, attribute dPdv, );

rtDeclareVariable (optix::Ray, ray, rtCurrentRay, );


static __device__ __inline__
void calc_uv()
{
    Dual2<Vec3> n (Vec3 (shading_normal.x, shading_normal.y, shading_normal.z));

    Dual2<float> nx(n.val().x, n.dx().x, n.dy().x);
    Dual2<float> ny(n.val().y, n.dx().y, n.dy().y);
    Dual2<float> nz(n.val().z, n.dx().z, n.dy().z);
    Dual2<float> u = (fast_atan2(nx, nz) + Dual2<float>(M_PI)) * 0.5f * float(M_1_PI);
    Dual2<float> v = fast_acos(ny) * float(M_1_PI);
    float xz2 = nx.val() * nx.val() + nz.val() * nz.val();
    if (xz2 > 0.0f) {
        const float PI = float(M_PI);
        const float TWOPI = float(2 * M_PI);
        float xz = sqrtf(xz2);
        float inv = 1.0f / xz;
        dPdu.x = -TWOPI * nx.val();
        dPdu.y = TWOPI * nz.val();
        dPdu.z = 0.0f;
        dPdv.x = -PI * nz.val() * inv * ny.val();
        dPdv.y = -PI * nx.val() * inv * ny.val();
        dPdv.z = PI * xz;
    } else {
        // pick arbitrary axes for poles to avoid division by 0
        if (ny.val() > 0.0f) {
            dPdu = make_float3 (0.0f, 0.0f, 1.0f);
            dPdv = make_float3 (1.0f, 0.0f, 0.0f);
        } else {
            dPdu = make_float3 ( 0.0f, 0.0f, 1.0f);
            dPdv = make_float3 (-1.0f, 0.0f, 0.0f);
        }
    }
    texcoord = make_float3 (u.val(), v.val(), 0.0f);
}


// Intersection adapted from testrender/raytracer.h
RT_PROGRAM void intersect (void)
{
    float3 c   = make_float3(sphere);
    float3 oc  = c - ray.origin;
    float  b   = dot(oc, ray.direction);
    float  det = b * b - dot(oc, oc) + r2;

    if (det >= 0.0f) {
        det = sqrtf(det);
        float x = b - det;
        float y = b + det;

        // NB: this does not included the 'self' check from
        // the testrender sphere intersection
        float t = (x > 0) ? x : ((y > 0) ? y : 0);

        if (rtPotentialIntersection(t)) {
            float3 P = ray.origin + ray.direction * t;
            float3 N = normalize (P - c);
            shading_normal = geometric_normal = N;
            surface_area = a;

            // Calcuate the texture coordinates and derivatives
            calc_uv();

            rtReportIntersection(0);
        }
    }
}


RT_PROGRAM void bounds (int, float result[6])
{
    const float3 center = make_float3(sphere);
    const float3 radius = make_float3(sphere.w);

    optix::Aabb* aabb = reinterpret_cast<optix::Aabb*>(result);

    if (radius.x > 0.0f && !isinf(radius.x)) {
        aabb->m_min = center - radius;
        aabb->m_max = center + radius;
    } else {
        aabb->invalidate();
    }
}

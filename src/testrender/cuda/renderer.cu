#include "hip/hip_runtime.h"
/*
Copyright (c) 2009-2018 Sony Pictures Imageworks Inc., et al.
All Rights Reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:
* Redistributions of source code must retain the above copyright
  notice, this list of conditions and the following disclaimer.
* Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions and the following disclaimer in the
  documentation and/or other materials provided with the distribution.
* Neither the name of Sony Pictures Imageworks nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_vector_types.h>

#include "util.h"


using namespace optix;

// Launch variables
rtDeclareVariable (uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable (uint2, launch_dim,   rtLaunchDim,   );

// Scene/Shading variables
rtDeclareVariable (float3,   bad_color, ,  );
rtDeclareVariable (float3,   bg_color, ,   );
rtDeclareVariable (rtObject, top_object, , );

// Ray payload
rtDeclareVariable (PRD_radiance, prd_radiance, rtPayload, );

// Geometry/Intersection attributes
rtDeclareVariable (float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable (float3, shading_normal,   attribute shading_normal, );

// Camera variables
rtDeclareVariable (float3, eye, , );
rtDeclareVariable (float3, dir, , );
rtDeclareVariable (float3, cx,  , );
rtDeclareVariable (float3, cy,  , );

rtDeclareVariable (float, invw, , );
rtDeclareVariable (float, invh, , );

// Buffers
rtBuffer<float3,2> output_buffer;


RT_PROGRAM void raygen()
{
    // Compute the pixel coordinates
    float2 d = make_float2 (static_cast<float>(launch_index.x) + 0.5f,
                            static_cast<float>(launch_index.y) + 0.5f);

    // Make the ray for the current pixel
    RayGeometry r;
    r.origin = eye;
    r.direction = optix::normalize(cx * (d.x * invw - 0.5f) + cy * (0.5f - d.y * invh) + dir);

    Ray ray = optix::make_Ray (r.origin, r.direction, 0, 1e-3f, RT_DEFAULT_MAX);

    // Create a struct to hold the shading result
    PRD_radiance prd;
    prd.result = make_float3 (0.0f);

    // Trace the ray against the scene. The hit/miss program is called before
    // this call returns.
    rtTrace (top_object, ray, prd);

    // Write the shading result to the output buffer
    output_buffer[launch_index] = prd.result;
}


RT_PROGRAM void miss()
{
    prd_radiance.result = bg_color;
}


RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
    output_buffer[launch_index] = bad_color;
}

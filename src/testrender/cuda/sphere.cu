#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/imageworks/OpenShadingLanguage


#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_vector_types.h>

#include <OSL/oslconfig.h>
#include <OSL/shaderglobals.h>
#include <OSL/dual.h>
#include <OSL/dual_vec.h>
#include <OpenImageIO/fmath.h>


using namespace optix;
using OSL::Dual2;
using OSL::Vec3;


rtDeclareVariable (float4, sphere, , );
rtDeclareVariable (float,  r2, , );
rtDeclareVariable (float,  a, , );

rtDeclareVariable (float3, texcoord,         attribute texcoord, );
rtDeclareVariable (float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable (float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable (float,  surface_area,     attribute surface_area, );

rtDeclareVariable (float3, dPdu, attribute dPdu, );
rtDeclareVariable (float3, dPdv, attribute dPdv, );

rtDeclareVariable (optix::Ray, ray, rtCurrentRay, );


static __device__ __inline__
void calc_uv()
{
    Dual2<Vec3> n (Vec3 (shading_normal.x, shading_normal.y, shading_normal.z));

    Dual2<float> nx(n.val().x, n.dx().x, n.dy().x);
    Dual2<float> ny(n.val().y, n.dx().y, n.dy().y);
    Dual2<float> nz(n.val().z, n.dx().z, n.dy().z);
    Dual2<float> u = (fast_atan2(nx, nz) + Dual2<float>(M_PI)) * 0.5f * float(M_1_PI);
    Dual2<float> v = fast_acos(ny) * float(M_1_PI);
    float xz2 = nx.val() * nx.val() + nz.val() * nz.val();
    if (xz2 > 0.0f) {
        const float PI = float(M_PI);
        const float TWOPI = float(2 * M_PI);
        float xz = sqrtf(xz2);
        float inv = 1.0f / xz;
        dPdu.x = -TWOPI * nx.val();
        dPdu.y = TWOPI * nz.val();
        dPdu.z = 0.0f;
        dPdv.x = -PI * nz.val() * inv * ny.val();
        dPdv.y = -PI * nx.val() * inv * ny.val();
        dPdv.z = PI * xz;
    } else {
        // pick arbitrary axes for poles to avoid division by 0
        if (ny.val() > 0.0f) {
            dPdu = make_float3 (0.0f, 0.0f, 1.0f);
            dPdv = make_float3 (1.0f, 0.0f, 0.0f);
        } else {
            dPdu = make_float3 ( 0.0f, 0.0f, 1.0f);
            dPdv = make_float3 (-1.0f, 0.0f, 0.0f);
        }
    }
    texcoord = make_float3 (u.val(), v.val(), 0.0f);
}


// Intersection adapted from testrender/raytracer.h
RT_PROGRAM void intersect (void)
{
    float3 c   = make_float3(sphere);
    float3 oc  = c - ray.origin;
    float  b   = dot(oc, ray.direction);
    float  det = b * b - dot(oc, oc) + r2;

    if (det >= 0.0f) {
        det = sqrtf(det);
        float x = b - det;
        float y = b + det;

        // NB: this does not included the 'self' check from
        // the testrender sphere intersection
        float t = (x > 0) ? x : ((y > 0) ? y : 0);

        if (rtPotentialIntersection(t)) {
            float3 P = ray.origin + ray.direction * t;
            float3 N = normalize (P - c);
            shading_normal = geometric_normal = N;
            surface_area = a;

            // Calcuate the texture coordinates and derivatives
            calc_uv();

            rtReportIntersection(0);
        }
    }
}


RT_PROGRAM void bounds (int, float result[6])
{
    const float3 center = make_float3(sphere);
    const float3 radius = make_float3(sphere.w);

    optix::Aabb* aabb = reinterpret_cast<optix::Aabb*>(result);

    if (radius.x > 0.0f && !isinf(radius.x)) {
        aabb->m_min = center - radius;
        aabb->m_max = center + radius;
    } else {
        aabb->invalidate();
    }
}

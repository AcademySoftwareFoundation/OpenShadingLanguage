#include "hip/hip_runtime.h"
/*
Copyright (c) 2009-2018 Sony Pictures Imageworks Inc., et al.
All Rights Reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:
* Redistributions of source code must retain the above copyright
  notice, this list of conditions and the following disclaimer.
* Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions and the following disclaimer in the
  documentation and/or other materials provided with the distribution.
* Neither the name of Sony Pictures Imageworks nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_vector_types.h>


using namespace optix;

rtDeclareVariable (float4, sphere, , );
rtDeclareVariable (float,  r2, , );

rtDeclareVariable (float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable (float3, shading_normal, attribute shading_normal, );
rtDeclareVariable (optix::Ray, ray, rtCurrentRay, );


// intersection adapted from testrender/raytracer.h
RT_PROGRAM void intersect (void)
{
    float3 c   = make_float3(sphere);
    float3 oc  = c - ray.origin;
    float  b   = dot(oc, ray.direction);
    float  det = b * b - dot(oc, oc) + r2;

    if (det >= 0.0f) {
        det = sqrtf(det);
        float x = b - det;
        float y = b + det;

        // NB: this does not included the 'self' check from
        // the testrender sphere intersection
        float t = (x > 0) ? x : ((y > 0) ? y : 0);

        if (rtPotentialIntersection(t)) {
            float3 P = ray.origin + ray.direction * t;
            float3 N = P - c;
            shading_normal = geometric_normal = N;
            rtReportIntersection(0);
        }
    }
}


RT_PROGRAM void bounds (int, float result[6])
{
    const float3 center = make_float3(sphere);
    const float3 radius = make_float3(sphere.w);

    optix::Aabb* aabb = reinterpret_cast<optix::Aabb*>(result);

    if (radius.x > 0.0f && !isinf(radius.x)) {
        aabb->m_min = center - radius;
        aabb->m_max = center + radius;
    } else {
        aabb->invalidate();
    }
}

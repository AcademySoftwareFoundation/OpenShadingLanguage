#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optix_math.h>

#include "rend_lib.h"


rtDeclareVariable (uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable (uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable (char*, test_str_1, , );
rtDeclareVariable (char*, test_str_2, , );


// These functions are declared extern to prevent name mangling.
extern "C" {

    __device__
    void* closure_component_allot (void* pool, int id, size_t prim_size, const float3& w)
    {
        ((ClosureComponent*) pool)->id = id;
        ((ClosureComponent*) pool)->w  = w;

        size_t needed   = (sizeof(ClosureComponent) - sizeof(void*) + prim_size + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return (void*) &char_ptr[needed];
    }


    __device__
    void* closure_mul_allot (void* pool, const float3& w, ClosureColor* c)
    {
        ((ClosureMul*) pool)->id      = ClosureColor::MUL;
        ((ClosureMul*) pool)->weight  = w;
        ((ClosureMul*) pool)->closure = c;

        size_t needed   = (sizeof(ClosureMul) + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return &char_ptr[needed];
    }


    __device__
    void* closure_mul_float_allot (void* pool, const float& w, ClosureColor* c)
    {
        ((ClosureMul*) pool)->id       = ClosureColor::MUL;
        ((ClosureMul*) pool)->weight.x = w;
        ((ClosureMul*) pool)->weight.y = w;
        ((ClosureMul*) pool)->weight.z = w;
        ((ClosureMul*) pool)->closure  = c;

        size_t needed   = (sizeof(ClosureMul) + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return &char_ptr[needed];
    }


    __device__
    void* closure_add_allot (void* pool, ClosureColor* a, ClosureColor* b)
    {
        ((ClosureAdd*) pool)->id       = ClosureColor::ADD;
        ((ClosureAdd*) pool)->closureA = a;
        ((ClosureAdd*) pool)->closureB = b;

        size_t needed   = (sizeof(ClosureAdd) + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return &char_ptr[needed];
    }


    __device__
    void* osl_allocate_closure_component (void* sg_, int id, int size)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        float3 w   = make_float3 (1.0f);
        void*  ret = sg_ptr->renderstate;

        size = max (4, size);

        sg_ptr->renderstate = closure_component_allot (sg_ptr->renderstate, id, size, w);

        return ret;
    }


    __device__
    void* osl_allocate_weighted_closure_component (void* sg_, int id, int size, const float3* w)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (w->x == 0.0f && w->y == 0.0f && w->z == 0.0f) {
            return NULL;
        }

        size = max (4, size);

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_component_allot (sg_ptr->renderstate, id, size, *w);

        return ret;
    }


    __device__
    void* osl_mul_closure_color (void* sg_, ClosureColor* a, float3* w)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (a == NULL) {
            return NULL;
        }

        if (w->x == 0.0f && w->y == 0.0f && w->z == 0.0f) {
            return NULL;
        }

        if (w->x == 1.0f && w->y == 1.0f && w->z == 1.0f) {
            return a;
        }

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_mul_allot (sg_ptr->renderstate, *w, a);

        return ret;
    }


    __device__
    void* osl_mul_closure_float (void* sg_, ClosureColor* a, float w)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (a == NULL || w == 0.0f) {
            return NULL;
        }

        if (w == 1.0f) {
            return a;
        }

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_mul_float_allot (sg_ptr->renderstate, w, a);

        return ret;
    }


    __device__
    void* osl_add_closure_closure (void* sg_, ClosureColor* a, ClosureColor* b)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (a == NULL) {
            return b;
        }

        if (b == NULL) {
            return a;
        }

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_add_allot (sg_ptr->renderstate, a, b);

        return ret;
    }


    __device__
    int rend_get_userdata (char* name, void* data, int data_size,
                           long long type, int index)
    {
        // Perform a userdata lookup using the parameter name, type, and
        // userdata index. If there is a match, memcpy the value into data and
        // return 1.

        // TODO: This is temporary code for initial testing and demonstration.
        if (IS_STRING(type) && HDSTR(name) == HDSTR(test_str_1)) {
            memcpy (data, &test_str_2, 8);
            return 1;
        }

        return 0;
    }


    __device__
    int osl_bind_interpolated_param (void *sg_, const void *name, long long type,
                                     int userdata_has_derivs, void *userdata_data,
                                     int symbol_has_derivs, void *symbol_data,
                                     int symbol_data_size,
                                     char *userdata_initialized, int userdata_index)
    {
        int status = rend_get_userdata ((char*)name, userdata_data, symbol_data_size,
                                        type, userdata_index);
        return status;
    }


    __device__
    int osl_strlen_is (const char *str)
    {
        return HDSTR(str).length();
    }


    __device__
    int osl_hash_is (const char *str)
    {
        return HDSTR(str).hash();
    }


    __device__
    int osl_getchar_isi (const char *str, int index)
    {
        return (str && unsigned(index) < HDSTR(str).length())
            ? str[index] : 0;
    }


    __device__
    void osl_printf (void* sg_, char* fmt_str, void* args)
    {
        printf (fmt_str, args);
    }


    __device__
    void* osl_get_noise_options (void *sg_)
    {
        ShaderGlobals* sg = ((ShaderGlobals*)sg_);
        NoiseOptCUDA* opt = (NoiseOptCUDA*)((ShadingContextCUDA*)sg->context)->noise_options_ptr();
        new (opt) NoiseOptCUDA;
        return opt;
    }


    __device__
    void* osl_get_texture_options (void *sg_)
    {
        return 0;
    }

    __device__
    void osl_texture_set_interp_code(void *opt, int mode)
    {
        // ((TextureOpt *)opt)->interpmode = (TextureOpt::InterpMode)mode;
    }

    __device__
    void osl_texture_set_stwrap_code (void *opt, int mode)
    {
        //((TextureOpt *)opt)->swrap = (TextureOpt::Wrap)mode;
        //((TextureOpt *)opt)->twrap = (TextureOpt::Wrap)mode;
    }

    __device__
    int osl_texture (void *sg_, const char *name, void *handle,
             void *opt_, float s, float t,
             float dsdx, float dtdx, float dsdy, float dtdy,
             int chans, void *result, void *dresultdx, void *dresultdy,
             void *alpha, void *dalphadx, void *dalphady,
             void *ustring_errormessage)
    {
        if (!handle)
            return 0;
        int64_t texID = int64_t(handle);
        *((float3*)result) = make_float3(optix::rtTex2D<float4>(texID, s, t));
        return 1;
    }

}

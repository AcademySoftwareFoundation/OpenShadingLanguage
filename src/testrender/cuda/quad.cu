#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/imageworks/OpenShadingLanguage


#include <optix.h>

#if (OPTIX_VERSION < 70000)
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_vector_types.h>


using namespace optix;

rtDeclareVariable (float3, p,  , );
rtDeclareVariable (float3, ex, , );
rtDeclareVariable (float3, ey, , );
rtDeclareVariable (float3, n,  , );
rtDeclareVariable (float,  eu, , );
rtDeclareVariable (float,  ev, , );
rtDeclareVariable (float,  a, ,  );

rtDeclareVariable (float3, texcoord,         attribute texcoord, );
rtDeclareVariable (float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable (float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable (float,  surface_area,     attribute surface_area, );

rtDeclareVariable (float3, dPdu, attribute dPdu, );
rtDeclareVariable (float3, dPdv, attribute dPdv, );

rtDeclareVariable (optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect (void)
{
    float dn = dot(ray.direction, n);
    float en = dot(p - ray.origin, n);
    if (dn * en > 0) {
        float  t  = en / dn;
        float3 h  = (ray.origin + ray.direction * t) - p;
        float  dx = dot(h, ex) * eu;
        float  dy = dot(h, ey) * ev;

        if (dx >= 0 && dx < 1.0f && dy >= 0 && dy < 1.0f && rtPotentialIntersection(t)) {
            shading_normal = geometric_normal = n;
            texcoord = make_float3(dot (h, ex) * eu, dot (h, ey) * ev, 0.0f);
            dPdu = ey;
            dPdv = ex;
            surface_area = a;
            rtReportIntersection(0);
        }
    }
}


RT_PROGRAM void bounds (int, float result[6])
{
    const float3 p00  = p;
    const float3 p01  = p + ex;
    const float3 p10  = p + ey;
    const float3 p11  = p + ex + ey;
    const float  area = length(cross(ex, ey));

    optix::Aabb* aabb = reinterpret_cast<optix::Aabb*>(result);

    if (area > 0.0f && !isinf(area)) {
        aabb->m_min = fminf (fminf (p00, p01), fminf (p10, p11));
        aabb->m_max = fmaxf (fmaxf (p00, p01), fmaxf (p10, p11));
    } else {
        aabb->invalidate();
    }
}

#else //#if (OPTIX_VERSION < 70000)

#include "wrapper.h"
#include "rend_lib.h"
#include "render_params.h"

extern "C" __device__
void __direct_callable__quad_shaderglobals (const unsigned int idx,
                                            const float        t_hit,
                                            const float3       ray_origin,
                                            const float3       ray_direction,
                                            ShaderGlobals     *sg)
{
    const GenericData *g_data  = reinterpret_cast<const GenericData *>(optixGetSbtDataPointer());
    const QuadParams *g_quads  = reinterpret_cast<const QuadParams *>(g_data->data);
    const QuadParams &quad     = g_quads[idx];
    const float3 P = ray_origin + t_hit * ray_direction;

    float3 h  = P - quad.p;

    sg->N = sg->Ng = quad.n;
    sg->u    = dot (h, quad.ex) * quad.eu;
    sg->v    = dot (h, quad.ey) * quad.ev;
    sg->dPdu = quad.ey;
    sg->dPdv = quad.ex;
    sg->surfacearea = quad.a;
    sg->shaderID    = quad.shaderID;
}


extern "C" __global__
void __intersection__quad ()
{
    const GenericData *g_data  = reinterpret_cast<const GenericData *>(optixGetSbtDataPointer());
    const QuadParams *g_quads  = reinterpret_cast<const QuadParams *>(g_data->data);
    const unsigned int idx     = optixGetPrimitiveIndex();
    const QuadParams &quad     = g_quads[idx];
    const float3 ray_origin    = optixGetObjectRayOrigin();
    const float3 ray_direction = optixGetObjectRayDirection();

    float dn = dot(ray_direction, quad.n);
    float en = dot(quad.p - ray_origin, quad.n);
    if (dn * en > 0) {
        float  t  = en / dn;
        float3 h  = (ray_origin + ray_direction * t) - quad.p;
        float  dx = dot(h, quad.ex) * quad.eu;
        float  dy = dot(h, quad.ey) * quad.ev;

        if (dx >= 0 && dx < 1.0f && dy >= 0 && dy < 1.0f && t < optixGetRayTmax())
            optixReportIntersection (t, RAYTRACER_HIT_QUAD);
    }
}

#endif //#if (OPTIX_VERSION < 70000)

#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/AcademySoftwareFoundation/OpenShadingLanguage

#include <OSL/hashes.h>
#include <optix.h>

#include <hip/hip_runtime.h>
#include <optix.h>

#include <OSL/oslclosure.h>

#include "optix_raytracer.h"
#include "rend_lib.h"
#include "vec_math.h"

#include "../render_params.h"
#include "../shading.h"

using OSL_CUDA::ShaderGlobals;


extern "C" {
__device__ __constant__ RenderParams render_params;
}


extern "C" __global__ void
__anyhit__any_hit_shadow()
{
    optixTerminateRay();
}



static __device__ void
globals_from_hit(OSL_CUDA::ShaderGlobals& sg)
{
    const GenericRecord* record = reinterpret_cast<GenericRecord*>(
        optixGetSbtDataPointer());

    OSL_CUDA::ShaderGlobals local_sg;
    // hit-kind 0: quad hit
    //          1: sphere hit
    optixDirectCall<void, unsigned int, float, float3, float3, OSL_CUDA::ShaderGlobals*>(
        optixGetHitKind(), optixGetPrimitiveIndex(), optixGetRayTmax(),
        optixGetWorldRayOrigin(), optixGetWorldRayDirection(), &local_sg);
    // Setup the ShaderGlobals
    const float3 ray_direction = optixGetWorldRayDirection();
    const float3 ray_origin    = optixGetWorldRayOrigin();
    const float t_hit          = optixGetRayTmax();

    sg.I  = ray_direction;
    sg.N  = normalize(optixTransformNormalFromObjectToWorldSpace(local_sg.N));
    sg.Ng = normalize(optixTransformNormalFromObjectToWorldSpace(local_sg.Ng));
    sg.P  = ray_origin + t_hit * ray_direction;
    sg.dPdu        = local_sg.dPdu;
    sg.dPdv        = local_sg.dPdv;
    sg.u           = local_sg.u;
    sg.v           = local_sg.v;
    sg.Ci          = NULL;
    sg.surfacearea = local_sg.surfacearea;
    sg.backfacing  = dot(sg.N, sg.I) > 0.0f;
    sg.shaderID    = local_sg.shaderID;

    if (sg.backfacing) {
        sg.N  = -sg.N;
        sg.Ng = -sg.Ng;
    }

    // NB: These variables are not used in the current iteration of the sample
    sg.raytype        = OSL::Ray::CAMERA;
    sg.flipHandedness = 0;
}



static __device__ float3
process_closure(const OSL::ClosureColor* closure_tree)
{
    OSL::Color3 result = OSL::Color3(0.0f);

    if (!closure_tree) {
        return make_float3(result.x, result.y, result.z);
    }

    // The depth of the closure tree must not exceed the stack size.
    // A stack size of 8 is probably quite generous for relatively
    // balanced trees.
    const int STACK_SIZE = 8;

    // Non-recursive traversal stack
    int stack_idx = 0;
    const OSL::ClosureColor* ptr_stack[STACK_SIZE];
    OSL::Color3 weight_stack[STACK_SIZE];

    // Shading accumulator
    OSL::Color3 weight = OSL::Color3(1.0f);

    const void* cur = closure_tree;
    while (cur) {
        ClosureIDs id = static_cast<ClosureIDs>(((OSL::ClosureColor*)cur)->id);
        switch (id) {
        case ClosureIDs::ADD: {
            ptr_stack[stack_idx]      = ((OSL::ClosureAdd*)cur)->closureB;
            weight_stack[stack_idx++] = weight;
            cur                       = ((OSL::ClosureAdd*)cur)->closureA;
            break;
        }

        case ClosureIDs::MUL: {
            weight *= ((OSL::ClosureMul*)cur)->weight;
            cur = ((OSL::ClosureMul*)cur)->closure;
            break;
        }

        case ClosureIDs::EMISSION_ID: {
            cur = NULL;
            break;
        }

        case ClosureIDs::DIFFUSE_ID:
        case ClosureIDs::OREN_NAYAR_ID:
        case ClosureIDs::PHONG_ID:
        case ClosureIDs::WARD_ID:
        case ClosureIDs::REFLECTION_ID:
        case ClosureIDs::REFRACTION_ID:
        case ClosureIDs::FRESNEL_REFLECTION_ID: {
            result += ((OSL::ClosureComponent*)cur)->w * weight;
            cur = NULL;
            break;
        }

        case ClosureIDs::MICROFACET_ID: {
            const char* mem = (const char*)((OSL::ClosureComponent*)cur)->data();
            OSL::ustringhash dist_uh = *(OSL::ustringhash*)&mem[0];

            if (dist_uh == OSL::Hashes::default_)
                return make_float3(0.0f, 1.0f, 1.0f);
            else
                return make_float3(1.0f, 0.0f, 1.0f);

            break;
        }

        default: cur = NULL; break;
        }

        if (cur == NULL && stack_idx > 0) {
            cur    = ptr_stack[--stack_idx];
            weight = weight_stack[stack_idx];
        }
    }

    return make_float3(result.x, result.y, result.z);
}



extern "C" __global__ void
__closesthit__closest_hit_osl()
{
    // TODO: Fixed-sized allocations can easily be exceeded by arbitrary shader
    //       networks, so there should be (at least) some mechanism to issue a
    //       warning or error if the closure or param storage can possibly be
    //       exceeded.
    alignas(8) char closure_pool[256];

    OSL_CUDA::ShaderGlobals sg;
    globals_from_hit(sg);

    // Pack the "closure pool" into one of the ShaderGlobals pointers
    *(int*)&closure_pool[0] = 0;
    sg.renderstate          = &closure_pool[0];

    // Create some run-time options structs. The OSL shader fills in the structs
    // as it executes, based on the options specified in the shader source.
    NoiseOptCUDA noiseopt;
    TextureOptCUDA textureopt;
    TraceOptCUDA traceopt;

    // Pack the pointers to the options structs in a faux "context",
    // which is a rough stand-in for the host ShadingContext.
    ShadingContextCUDA shading_context = { &noiseopt, &textureopt, &traceopt };

    sg.context = &shading_context;

    // Run the OSL callable
    void* interactive_ptr = reinterpret_cast<void**>(
        render_params.interactive_params)[sg.shaderID];
    const unsigned int shaderIdx = 2u + sg.shaderID + 0u;
    optixDirectCall<void, OSL_CUDA::ShaderGlobals*, void*, void*, void*, int, void*>(
        shaderIdx, &sg /*shaderglobals_ptr*/, nullptr /*groupdata_ptr*/,
        nullptr /*userdata_base_ptr*/, nullptr /*output_base_ptr*/,
        0 /*shadeindex - unused*/, interactive_ptr /*interactive_params_ptr*/);

    float3 result      = process_closure((OSL::ClosureColor*)sg.Ci);
    uint3 launch_dims  = optixGetLaunchDimensions();
    uint3 launch_index = optixGetLaunchIndex();

    float3* output_buffer = reinterpret_cast<float3*>(
        render_params.output_buffer);
    int pixel            = launch_index.y * launch_dims.x + launch_index.x;
    output_buffer[pixel] = make_float3(result.x, result.y, result.z);
}

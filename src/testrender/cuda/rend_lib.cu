#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/AcademySoftwareFoundation/OpenShadingLanguage

#include <optix.h>
#include <optix_device.h>

#define OPTIX_COMPATIBILITY 7
#include <OSL/oslclosure.h>

#include <hip/hip_runtime.h>
#include <optix_device.h>

#include "rend_lib.h"

#define MEMCPY_ALIGNED(dst, src, size, alignment)    \
    memcpy(__builtin_assume_aligned(dst, alignment), \
           __builtin_assume_aligned(src, alignment), size);

OSL_NAMESPACE_BEGIN
namespace pvt {
extern __device__ hipDeviceptr_t s_color_system;
extern __device__ hipDeviceptr_t osl_printf_buffer_start;
extern __device__ hipDeviceptr_t osl_printf_buffer_end;
extern __device__ uint64_t test_str_1;
extern __device__ uint64_t test_str_2;
extern __device__ uint64_t num_named_xforms;
extern __device__ hipDeviceptr_t xform_name_buffer;
extern __device__ hipDeviceptr_t xform_buffer;
}  // namespace pvt
OSL_NAMESPACE_END


// Taken from the SimplePool class
__device__ static inline size_t
alignment_offset_calc(void* ptr, size_t alignment)
{
    uintptr_t ptrbits = reinterpret_cast<uintptr_t>(ptr);
    uintptr_t offset  = ((ptrbits + alignment - 1) & -alignment) - ptrbits;
    return offset;
}


// These functions are declared extern to prevent name mangling.
extern "C" {

// add OptiX entry point to prevent OptiX from discarding the module
__global__ void
__direct_callable__dummy_rend_lib()
{
}


__device__ void*
closure_component_allot(void* pool, int id, size_t prim_size,
                        const OSL::Color3& w)
{
    ((OSL::ClosureComponent*)pool)->id = id;
    ((OSL::ClosureComponent*)pool)->w  = w;

    size_t needed = (sizeof(OSL::ClosureComponent) + prim_size
                     + (alignof(OSL::ClosureComponent) - 1))
                    & ~(alignof(OSL::ClosureComponent) - 1);
    char* char_ptr = (char*)pool;

    return (void*)&char_ptr[needed];
}


__device__ void*
closure_mul_allot(void* pool, const OSL::Color3& w, OSL::ClosureColor* c)
{
    ((OSL::ClosureMul*)pool)->id      = OSL::ClosureColor::MUL;
    ((OSL::ClosureMul*)pool)->weight  = w;
    ((OSL::ClosureMul*)pool)->closure = c;

    size_t needed = (sizeof(OSL::ClosureMul)
                     + (alignof(OSL::ClosureComponent) - 1))
                    & ~(alignof(OSL::ClosureComponent) - 1);
    char* char_ptr = (char*)pool;

    return &char_ptr[needed];
}


__device__ void*
closure_mul_float_allot(void* pool, const float& w, OSL::ClosureColor* c)
{
    ((OSL::ClosureMul*)pool)->id       = OSL::ClosureColor::MUL;
    ((OSL::ClosureMul*)pool)->weight.x = w;
    ((OSL::ClosureMul*)pool)->weight.y = w;
    ((OSL::ClosureMul*)pool)->weight.z = w;
    ((OSL::ClosureMul*)pool)->closure  = c;

    size_t needed = (sizeof(OSL::ClosureMul)
                     + (alignof(OSL::ClosureComponent) - 1))
                    & ~(alignof(OSL::ClosureComponent) - 1);
    char* char_ptr = (char*)pool;

    return &char_ptr[needed];
}


__device__ void*
closure_add_allot(void* pool, OSL::ClosureColor* a, OSL::ClosureColor* b)
{
    ((OSL::ClosureAdd*)pool)->id       = OSL::ClosureColor::ADD;
    ((OSL::ClosureAdd*)pool)->closureA = a;
    ((OSL::ClosureAdd*)pool)->closureB = b;

    size_t needed = (sizeof(OSL::ClosureAdd)
                     + (alignof(OSL::ClosureComponent) - 1))
                    & ~(alignof(OSL::ClosureComponent) - 1);
    char* char_ptr = (char*)pool;

    return &char_ptr[needed];
}


__device__ void*
osl_allocate_closure_component(void* sg_, int id, int size)
{
    OSL_CUDA::ShaderGlobals* sg_ptr = (OSL_CUDA::ShaderGlobals*)sg_;

    OSL::Color3 w = OSL::Color3(1, 1, 1);
    // Fix up the alignment
    void* ret = ((char*)sg_ptr->renderstate)
                + alignment_offset_calc(sg_ptr->renderstate,
                                        alignof(OSL::ClosureComponent));

    size = max(4, size);

    sg_ptr->renderstate = closure_component_allot(ret, id, size, w);

    return ret;
}



__device__ void*
osl_allocate_weighted_closure_component(void* sg_, int id, int size,
                                        const void* w)
{
    OSL_CUDA::ShaderGlobals* sg_ptr = (OSL_CUDA::ShaderGlobals*)sg_;

    const OSL::Color3* wc
        = (const OSL::Color3*)__builtin_assume_aligned(w, alignof(float));

    if (wc->x == 0.0f && wc->y == 0.0f && wc->z == 0.0f) {
        return NULL;
    }

    size = max(4, size);

    // Fix up the alignment
    void* ret = ((char*)sg_ptr->renderstate)
                + alignment_offset_calc(sg_ptr->renderstate,
                                        alignof(OSL::ClosureComponent));
    sg_ptr->renderstate = closure_component_allot(ret, id, size, *wc);

    return ret;
}



__device__ void*
osl_mul_closure_color(void* sg_, void* a, const void* w)
{
    OSL_CUDA::ShaderGlobals* sg_ptr = (OSL_CUDA::ShaderGlobals*)sg_;
    const OSL::Color3* wc
        = (const OSL::Color3*)__builtin_assume_aligned(w, alignof(float));

    if (a == NULL) {
        return NULL;
    }

    if (wc->x == 0.0f && wc->y == 0.0f && wc->z == 0.0f) {
        return NULL;
    }

    if (wc->x == 1.0f && wc->y == 1.0f && wc->z == 1.0f) {
        return a;
    }

    // Fix up the alignment
    void* ret = ((char*)sg_ptr->renderstate)
                + alignment_offset_calc(sg_ptr->renderstate,
                                        alignof(OSL::ClosureComponent));
    sg_ptr->renderstate = closure_mul_allot(ret, *wc, (OSL::ClosureColor*)a);

    return ret;
}



__device__ void*
osl_mul_closure_float(void* sg_, void* a, float w)
{
    a = __builtin_assume_aligned(a, alignof(float));

    OSL_CUDA::ShaderGlobals* sg_ptr = (OSL_CUDA::ShaderGlobals*)sg_;

    if (a == NULL || w == 0.0f) {
        return NULL;
    }

    if (w == 1.0f) {
        return a;
    }

    // Fix up the alignment
    void* ret = ((char*)sg_ptr->renderstate)
                + alignment_offset_calc(sg_ptr->renderstate,
                                        alignof(OSL::ClosureComponent));
    sg_ptr->renderstate = closure_mul_float_allot(ret, w,
                                                  (OSL::ClosureColor*)a);

    return ret;
}



__device__ void*
osl_add_closure_closure(void* sg_, void* a, void* b)
{
    a = __builtin_assume_aligned(a, alignof(float));
    b = __builtin_assume_aligned(b, alignof(float));

    OSL_CUDA::ShaderGlobals* sg_ptr = (OSL_CUDA::ShaderGlobals*)sg_;

    if (a == NULL) {
        return b;
    }

    if (b == NULL) {
        return a;
    }

    // Fix up the alignment
    void* ret = ((char*)sg_ptr->renderstate)
                + alignment_offset_calc(sg_ptr->renderstate,
                                        alignof(OSL::ClosureComponent));
    sg_ptr->renderstate = closure_add_allot(ret, (OSL::ClosureColor*)a,
                                            (OSL::ClosureColor*)b);

    return ret;
}


#define IS_STRING(type) (type.basetype == OSL::TypeDesc::STRING)
#define IS_PTR(type)    (type.basetype == OSL::TypeDesc::PTR)
#define IS_COLOR(type)  (type.vecsemantics == OSL::TypeDesc::COLOR)


__device__ bool
rend_get_userdata(OSL::ustringhash name, void* data, int data_size,
                  const OSL::TypeDesc& type, int index)
{
    // Perform a userdata lookup using the parameter name, type, and
    // userdata index. If there is a match, memcpy the value into data and
    // return 1.

    if (IS_PTR(type) && name == OSL::Hashes::colorsystem) {
        *(void**)data = *reinterpret_cast<void**>(&OSL::pvt::s_color_system);
        return true;
    }
    // TODO: This is temporary code for initial testing and demonstration.
    if (IS_STRING(type) && name == HDSTR(OSL::pvt::test_str_1)) {
        MEMCPY_ALIGNED(data, &OSL::pvt::test_str_2, 8, alignof(float));
        return true;
    }

    return false;
}

#undef IS_COLOR
#undef IS_STRING
#undef IS_PTR


__device__ int
osl_bind_interpolated_param(void* sg_, OSL::ustringhash_pod name_,
                            long long type, int userdata_has_derivs,
                            void* userdata_data, int symbol_has_derivs,
                            void* symbol_data, int symbol_data_size,
                            char* userdata_initialized, int userdata_index)
{
    char status = *userdata_initialized;
    if (status == 0) {
        OSL::ustringhash name = OSL::ustringhash_from(name_);
        bool ok = rend_get_userdata(name, userdata_data, symbol_data_size,
                                    (*(OSL::TypeDesc*)&type), userdata_index);
        *userdata_initialized = status = 1 + ok;
    }
    if (status == 2) {
        MEMCPY_ALIGNED(symbol_data, userdata_data, symbol_data_size,
                       alignof(float));
        return 1;
    }
    return 0;
}


__device__ int
osl_strlen_is(OSL::ustringhash_pod str)
{
    //return HDSTR(str).length();
    return 0;
}


__device__ int
osl_hash_is(OSL::ustringhash_pod str)
{
    return static_cast<int>(str);
}


__device__ int
osl_getchar_isi(OSL::ustringhash_pod str, int index)
{
    //        return (str && unsigned(index) < HDSTR(str).length())
    //            ? str[index] : 0;
    return 0;
}



// Printing is handled by the host.  Copy format string's hash and
// all the arguments to our print buffer.
// Note:  the first element of 'args' is the size of the argument list
__device__ void
osl_printf(void* sg_, OSL::ustringhash_pod fmt_str_hash, void* args)
{
    uint64_t args_size = reinterpret_cast<uint64_t*>(args)[0];

    // This can be used to limit printing to one Cuda thread for debugging
    // if (launch_index.x == 0 && launch_index.y == 0)

    hipDeviceptr_t copy_start = atomicAdd(&OSL::pvt::osl_printf_buffer_start,
                                       args_size + sizeof(args_size)
                                           + sizeof(fmt_str_hash));

    // Only perform copy if there's enough space
    if (copy_start + args_size + sizeof(args_size) + sizeof(fmt_str_hash)
        < OSL::pvt::osl_printf_buffer_end) {
        memcpy(reinterpret_cast<void*>(copy_start), &fmt_str_hash,
               sizeof(fmt_str_hash));
        memcpy(reinterpret_cast<void*>(copy_start + sizeof(fmt_str_hash)),
               &args_size, sizeof(args_size));
        memcpy(reinterpret_cast<void*>(copy_start + sizeof(fmt_str_hash)
                                       + sizeof(args_size)),
               reinterpret_cast<char*>(args) + sizeof(args_size), args_size);
    }
}



__forceinline__ __device__ float3
make_float3(const float4& a)
{
    return make_float3(a.x, a.y, a.z);
}



__device__ int
osl_range_check_err(int indexvalue, int length, OSL::ustringhash_pod symname,
                    void* sg, OSL::ustringhash_pod sourcefile, int sourceline,
                    OSL::ustringhash_pod groupname, int layer,
                    OSL::ustringhash_pod layername,
                    OSL::ustringhash_pod shadername)
{
    if (indexvalue < 0 || indexvalue >= length) {
        return indexvalue < 0 ? 0 : length - 1;
    }
    return indexvalue;
}


#define MAT(m) (*(OSL::Matrix44*)__builtin_assume_aligned(m, alignof(float)))

__device__ int
osl_get_matrix(void* sg_, void* r, OSL::ustringhash_pod from_)
{
    r                           = __builtin_assume_aligned(r, alignof(float));
    OSL::ustringhash from       = OSL::ustringhash_from(from_);
    OSL_CUDA::ShaderGlobals* sg = (OSL_CUDA::ShaderGlobals*)sg_;
    if (from == OSL::Hashes::common) {
        MAT(r).makeIdentity();
        return true;
    }
    if (from == OSL::Hashes::object) {
        MAT(r) = MAT(sg->object2common);
        return true;
    }
    if (from == OSL::Hashes::shader) {
        MAT(r) = MAT(sg->shader2common);
        return true;
    }

    // Find the index of the named transform in the transform list
    int match_idx = -1;
    for (size_t idx = 0; idx < OSL::pvt::num_named_xforms; ++idx) {
        if (from == HDSTR(((uint64_t*)OSL::pvt::xform_name_buffer)[idx])) {
            match_idx = static_cast<int>(idx);
            break;
        }
    }

    // Return the transform if there is a match
    if (match_idx >= 0) {
        MAT(r) = reinterpret_cast<OSL::Matrix44*>(
            OSL::pvt::xform_buffer)[match_idx];
        return true;
    }

    int ok = false;  // TODO: Implement transform
    if (!ok) {
        MAT(r).makeIdentity();
        // TBR: OSL would throw an error here, what should we do?
    }
    return ok;
}



__device__ int
osl_get_inverse_matrix(void* sg_, void* r, OSL::ustringhash_pod to_)
{
    r                           = __builtin_assume_aligned(r, alignof(float));
    OSL::ustringhash to         = OSL::ustringhash_from(to_);
    OSL_CUDA::ShaderGlobals* sg = (OSL_CUDA::ShaderGlobals*)sg_;
    if (to == OSL::Hashes::common) {
        MAT(r).makeIdentity();
        return true;
    }
    if (to == OSL::Hashes::object) {
        MAT(r) = MAT(sg->object2common);
        MAT(r).invert();
        return true;
    }
    if (to == OSL::Hashes::shader) {
        MAT(r) = MAT(sg->shader2common);
        MAT(r).invert();
        return true;
    }

    // Find the index of the named transform in the transform list
    int match_idx = -1;
    for (size_t idx = 0; idx < OSL::pvt::num_named_xforms; ++idx) {
        if (to == HDSTR(((uint64_t*)OSL::pvt::xform_name_buffer)[idx])) {
            match_idx = static_cast<int>(idx);
            break;
        }
    }
    // Return the transform if there is a match
    if (match_idx >= 0) {
        MAT(r) = reinterpret_cast<OSL::Matrix44*>(
            OSL::pvt::xform_buffer)[match_idx];
        MAT(r).invert();
        return true;
    }

    int ok = false;  // TODO: Implement transform
    if (!ok) {
        MAT(r).makeIdentity();
        // TBR: OSL would throw an error here, what should we do?
    }
    return ok;
}

#undef MAT
}

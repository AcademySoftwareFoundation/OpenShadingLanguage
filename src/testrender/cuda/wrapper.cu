#include "hip/hip_runtime.h"
// Copyright Contributors to the Open Shading Language project.
// SPDX-License-Identifier: BSD-3-Clause
// https://github.com/imageworks/OpenShadingLanguage


#include <optix.h>

#if (OPTIX_VERSION < 70000) 
#include <optixu/optixu_math_namespace.h>
#else
#include <optix.h>
#include <hip/hip_runtime.h>
#endif

#include <OSL/device_string.h>
#include <OSL/oslclosure.h>

#include "rend_lib.h"
#include "util.h"

#if (OPTIX_VERSION < 70000)

// Ray payload
rtDeclareVariable (PRD_radiance, prd_radiance, rtPayload, );

// ray/hit variables
rtDeclareVariable (float3, shading_normal,   attribute shading_normal,  );
rtDeclareVariable (float3, geometric_normal, attribute geometric_normal,);
rtDeclareVariable (float3, texcoord,         attribute texcoord,        );
rtDeclareVariable (float,  surface_area,     attribute surface_area,    );
rtDeclareVariable (float3, dPdu,             attribute dPdu,            );
rtDeclareVariable (float3, dPdv,             attribute dPdv,            );
rtDeclareVariable (int,    obj_id,           attribute obj_id,          );
rtDeclareVariable (int,    lgt_idx,          attribute lgt_idx,         );

// ray/hit variables
rtDeclareVariable (uint2,      launch_index, rtLaunchIndex,          );
rtDeclareVariable (uint2,      launch_dim,   rtLaunchDim,            );
rtDeclareVariable (optix::Ray, ray,          rtCurrentRay,           );
rtDeclareVariable (float,      t_hit,        rtIntersectionDistance, );

// Buffers
rtBuffer<float3,2> output_buffer;

// Function pointers for the OSL shader
rtDeclareVariable (rtCallableProgramId<void (void*, void*)>, osl_init_func, , );
rtDeclareVariable (rtCallableProgramId<void (void*, void*)>, osl_group_func, ,);

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}


static __device__
void globals_from_hit(ShaderGlobals& sg)
{
    // Setup the ShaderGlobals
    sg.I           = ray.direction;
    sg.N           = normalize(rtTransformNormal (RT_OBJECT_TO_WORLD, shading_normal));
    sg.Ng          = normalize(rtTransformNormal (RT_OBJECT_TO_WORLD, geometric_normal));
    sg.P           = ray.origin + t_hit * ray.direction;
    sg.dPdu        = dPdu;
    sg.u           = texcoord.x;
    sg.v           = texcoord.y;
    sg.Ci          = NULL;
    sg.surfacearea = surface_area;
    sg.backfacing  = (dot(sg.N, sg.I) > 0.0f);

    if (sg.backfacing) {
        sg.N  = -sg.N;
        sg.Ng = -sg.Ng;
    }

    // NB: These variables are not used in the current iteration of the sample
    sg.raytype = CAMERA;
    sg.flipHandedness = 0;
}


static __device__
float3 process_closure(const OSL::ClosureColor* closure_tree)
{
    OSL::Color3 result = OSL::Color3 (0.0f);

    if (!closure_tree) {
        return make_float3(result.x, result.y, result.z);
    }

    // The depth of the closure tree must not exceed the stack size.
    // A stack size of 8 is probably quite generous for relatively
    // balanced trees.
    const int STACK_SIZE = 8;

    // Non-recursive traversal stack
    int    stack_idx = 0;
    const OSL::ClosureColor* ptr_stack[STACK_SIZE];
    OSL::Color3 weight_stack[STACK_SIZE];

    // Shading accumulator
    OSL::Color3 weight = OSL::Color3(1.0f);

    const void* cur = closure_tree;
    while (cur) {
        switch (((OSL::ClosureColor*)cur)->id) {
        case OSL::ClosureColor::ADD: {
            ptr_stack   [stack_idx  ] = ((OSL::ClosureAdd*) cur)->closureB;
            weight_stack[stack_idx++] = weight;
            cur = ((OSL::ClosureAdd*) cur)->closureA;
            break;
        }

        case OSL::ClosureColor::MUL: {
            weight *= ((OSL::ClosureMul*) cur)->weight;
            cur     = ((OSL::ClosureMul*) cur)->closure;
            break;
        }

        case EMISSION_ID: {
            cur = NULL;
            break;
        }

        case DIFFUSE_ID:
        case OREN_NAYAR_ID:
        case PHONG_ID:
        case WARD_ID:
        case REFLECTION_ID:
        case REFRACTION_ID:
        case FRESNEL_REFLECTION_ID: {
            result += ((OSL::ClosureComponent*) cur)->w * weight;
            cur = NULL;
            break;
        }

        case MICROFACET_ID: {
            const char* mem = (const char*)((OSL::ClosureComponent*) cur)->data();
            const char* dist_str = *(const char**) &mem[0];
#if 0
            if (launch_index.x == launch_dim.x / 2 && launch_index.y == launch_dim.y / 2)
                printf ("microfacet, dist: %s\n", HDSTR(dist_str).c_str());
#endif

            if (HDSTR(dist_str) == OSL::DeviceStrings::default_)
                return make_float3(0.0f, 1.0f, 1.0f);

            return make_float3(1.0f, 0.0f, 1.0f);
        }

        default:
            cur = NULL;
            break;
        }

        if (cur == NULL && stack_idx > 0) {
            cur    = ptr_stack   [--stack_idx];
            weight = weight_stack[  stack_idx];
        }
    }

    return make_float3(result.x, result.y, result.z);
}


RT_PROGRAM void closest_hit_osl()
{
    // TODO: Fixed-sized allocations can easily be exceeded by arbitrary shader
    //       networks, so there should be (at least) some mechanism to issue a
    //       warning or error if the closure or param storage can possibly be
    //       exceeded.
    alignas(8) char closure_pool[256];
    alignas(8) char params      [256];

    ShaderGlobals sg;
    globals_from_hit (sg);

    // Pack the "closure pool" into one of the ShaderGlobals pointers
    *(int*) &closure_pool[0] = 0;
    sg.renderstate = &closure_pool[0];

    // Create some run-time options structs. The OSL shader fills in the structs
    // as it executes, based on the options specified in the shader source.
    NoiseOptCUDA   noiseopt;
    TextureOptCUDA textureopt;
    TraceOptCUDA   traceopt;

    // Pack the pointers to the options structs in a faux "context",
    // which is a rough stand-in for the host ShadingContext.
    ShadingContextCUDA shading_context = {
        &noiseopt, &textureopt, &traceopt
    };

    sg.context = &shading_context;

    // Run the OSL group and init functions
    osl_init_func (&sg, params);
    osl_group_func(&sg, params);

    prd_radiance.result = process_closure ((OSL::ClosureColor*) sg.Ci);
}

#else //#if (OPTIX_VERSION < 70000)


#include "../render_params.h"

extern "C" {
__device__ __constant__ RenderParams render_params;
}

extern"C" __global__ void __anyhit__any_hit_shadow ()
{
    optixTerminateRay();
}


static __device__
void globals_from_hit (ShaderGlobals& sg)
{
    const GenericRecord *record = reinterpret_cast<GenericRecord *> (optixGetSbtDataPointer());

    ShaderGlobals local_sg;
    // hit-kind 0: quad hit
    //          1: sphere hit
    optixDirectCall<void, unsigned int, float, float3, float3, ShaderGlobals *>(
                                     optixGetHitKind(),
                                     optixGetPrimitiveIndex(),
                                     optixGetRayTmax(),
                                     optixGetWorldRayOrigin(),
                                     optixGetWorldRayDirection(),
                                     &local_sg);
    // Setup the ShaderGlobals
    const float3 ray_direction = optixGetWorldRayDirection();
    const float3 ray_origin    = optixGetWorldRayOrigin();
    const float  t_hit         = optixGetRayTmin();

    sg.I           = ray_direction;
    sg.N           = normalize (optixTransformNormalFromObjectToWorldSpace (local_sg.N));
    sg.Ng          = normalize (optixTransformNormalFromObjectToWorldSpace (local_sg.Ng));
    sg.P           = ray_origin + t_hit * ray_direction;
    sg.dPdu        = local_sg.dPdu;
    sg.dPdv        = local_sg.dPdv;
    sg.u           = local_sg.u;
    sg.v           = local_sg.v;
    sg.Ci          = NULL;
    sg.surfacearea = local_sg.surfacearea;
    sg.backfacing  = dot (sg.N, sg.I) > 0.0f;
    sg.shaderID    = local_sg.shaderID;

    if (sg.backfacing) {
        sg.N  = -sg.N;
        sg.Ng = -sg.Ng;
    }

    // NB: These variables are not used in the current iteration of the sample
    sg.raytype = CAMERA;
    sg.flipHandedness = 0;
}


static __device__
float3 process_closure (const OSL::ClosureColor* closure_tree)
{
    OSL::Color3 result = OSL::Color3 (0.0f);

    if (!closure_tree) {
        return make_float3 (result.x, result.y, result.z);
    }

    // The depth of the closure tree must not exceed the stack size.
    // A stack size of 8 is probably quite generous for relatively
    // balanced trees.
    const int STACK_SIZE = 8;

    // Non-recursive traversal stack
    int    stack_idx = 0;
    const OSL::ClosureColor* ptr_stack[STACK_SIZE];
    OSL::Color3 weight_stack[STACK_SIZE];

    // Shading accumulator
    OSL::Color3 weight = OSL::Color3 (1.0f);

    const void* cur = closure_tree;
    while (cur) {
        switch (((OSL::ClosureColor*)cur)->id) {
        case OSL::ClosureColor::ADD: {
            ptr_stack   [stack_idx  ] = ((OSL::ClosureAdd*) cur)->closureB;
            weight_stack[stack_idx++] = weight;
            cur = ((OSL::ClosureAdd*) cur)->closureA;
            break;
        }

        case OSL::ClosureColor::MUL: {
            weight *= ((OSL::ClosureMul*) cur)->weight;
            cur     = ((OSL::ClosureMul*) cur)->closure;
            break;
        }

        case EMISSION_ID: {
            cur = NULL;
            break;
        }

        case DIFFUSE_ID:
        case OREN_NAYAR_ID:
        case PHONG_ID:
        case WARD_ID:
        case REFLECTION_ID:
        case REFRACTION_ID:
        case FRESNEL_REFLECTION_ID: {
            result += ((OSL::ClosureComponent*) cur)->w * weight;
            cur = NULL;
            break;
        }

        case MICROFACET_ID: {
            const char* mem = (const char*)((OSL::ClosureComponent*) cur)->data();
            const char* dist_str = *(const char**) &mem[0];

            if (HDSTR(dist_str) == OSL::DeviceStrings::default_)
                return make_float3(0.0f, 1.0f, 1.0f);
            else
                return make_float3(1.0f, 0.0f, 1.0f);

            break;
        }

        default:
            cur = NULL;
            break;
        }

        if (cur == NULL && stack_idx > 0) {
            cur    = ptr_stack   [--stack_idx];
            weight = weight_stack[  stack_idx];
        }
    }

    return make_float3(result.x, result.y, result.z);
}


extern "C" __global__  void __closesthit__closest_hit_osl()
{
    // TODO: Fixed-sized allocations can easily be exceeded by arbitrary shader
    //       networks, so there should be (at least) some mechanism to issue a
    //       warning or error if the closure or param storage can possibly be
    //       exceeded.
    alignas(8) char closure_pool[256];
    alignas(8) char params      [256];

    ShaderGlobals sg;
    globals_from_hit (sg);

    // Pack the "closure pool" into one of the ShaderGlobals pointers
    *(int*) &closure_pool[0] = 0;
    sg.renderstate = &closure_pool[0];

    // Create some run-time options structs. The OSL shader fills in the structs
    // as it executes, based on the options specified in the shader source.
    NoiseOptCUDA   noiseopt;
    TextureOptCUDA textureopt;
    TraceOptCUDA   traceopt;

    // Pack the pointers to the options structs in a faux "context",
    // which is a rough stand-in for the host ShadingContext.
    ShadingContextCUDA shading_context = {
        &noiseopt, &textureopt, &traceopt
    };

    sg.context = &shading_context;

    // Run the OSL group and init functions
    const unsigned int shaderInitOpIdx = 2u + 2u * sg.shaderID + 0u;
    const unsigned int shaderGroupIdx  = 2u + 2u * sg.shaderID + 1u;
    optixDirectCall<void, ShaderGlobals*, void *>(shaderInitOpIdx, &sg, params); // call osl_init_func
    optixDirectCall<void, ShaderGlobals*, void *>(shaderGroupIdx , &sg, params); // call osl_group_func

    float3 result = process_closure ((OSL::ClosureColor*) sg.Ci);
    uint3 launch_dims  = optixGetLaunchDimensions();
    uint3 launch_index = optixGetLaunchIndex();

    float3* output_buffer = reinterpret_cast<float3 *>(render_params.output_buffer);
    int pixel = launch_index.y * launch_dims.x + launch_index.x;
    output_buffer[pixel] = make_float3(result.x, result.y, result.z);

}

#endif //#if (OPTIX_VERSION < 70000)
